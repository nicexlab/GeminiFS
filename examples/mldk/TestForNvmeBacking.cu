#include <cassert>
#include <iostream>
#include <ctime>
#include "linux/ioctl.h"
#include "geminifs_api.h"

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer() {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer() {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start() {
            hipEventRecord(start, 0);
      }

      void Stop() {
            hipEventRecord(stop, 0);
      }

      float Elapsed() {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define my_assert(code) do { \
    if (!(code)) { \
        host_close_all(); \
        exit(1); \
    } \
} while(0)


#define snvme_control_path "/dev/snvm_control"
#define snvme_path "/dev/csnvme1"
#define nvme_dev_path "/dev/snvme0n1"
#define snvme_helper_path "/dev/snvme_helper"
#define nvme_mount_path "/mnt/nvm_mount"
#define nvme_pci_addr {0xc3, 0, 0}

#define geminifs_file_name "checkpoint.geminifs"
#define geminifs_file_path (nvme_mount_path "/" geminifs_file_name)

#define NR_WARPS 1
#define NR_PAGES__PER_WARP 1

int
main() {
    host_open_all(
            snvme_control_path,
            snvme_path,
            nvme_dev_path,
            nvme_mount_path,
            nvme_pci_addr,
            1,
            1024,
            64);

    int nr_warps = NR_WARPS;

  size_t file_block_size = 4 * (1ull << 10);
  size_t dev_page_size = 128 * (1ull << 10);


  size_t nr_pages = nr_warps * NR_PAGES__PER_WARP;
  size_t page_capacity = nr_pages * dev_page_size;
  size_t virtual_space_size = page_capacity * 8;

  srand(time(0));
  int rand_start = rand();

  remove(geminifs_file_path);

  host_fd_t host_fd = host_create_geminifs_file(geminifs_file_path, file_block_size, virtual_space_size);
  host_refine_nvmeofst(host_fd);

  uint64_t *buf1 = (uint64_t *)malloc(virtual_space_size);
  for (size_t i = 0; i < virtual_space_size / sizeof(uint64_t); i++)
      buf1[i] = rand_start + i;
  host_xfer_geminifs_file(host_fd, 0, buf1, virtual_space_size, 0);
  
  uint64_t *buf2 = (uint64_t *)malloc(virtual_space_size);
  // todo 
  host_xfer_geminifs_file(host_fd, 0, buf2, virtual_space_size, 1);
  for (size_t i = 0; i < virtual_space_size / sizeof(uint64_t); i++)
      my_assert(buf2[i] == rand_start + i);

  host_close_geminifs_file(host_fd);

  printf("ALL OK!\n");
  host_close_all();

  return 0;
}

